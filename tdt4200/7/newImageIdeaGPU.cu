#include "hip/hip_runtime.h"
#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "ppmCU.h"

// Image from:
// http://7-themes.com/6971875-funny-flowers-pictures.html

typedef struct {
     float red,green,blue;
} AccuratePixel;

typedef struct {
     int x, y;
     AccuratePixel *data;
} AccurateImage;

__global__ void convertImageToNewFormatGPU(float* imageUnchanged, unsigned char* image, int W, int H)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H) {
        return;
    }
    int i = (W * y + x) * 3;
    imageUnchanged[i]   = __uint2float_rd(image[i]);
    imageUnchanged[i + 1] = __uint2float_rd(image[i + 1]);
    imageUnchanged[i + 2] = __uint2float_rd(image[i + 2]);
}

// free memory of an AccurateImage
void freeImage(AccurateImage *image){
	free(image->data);
	free(image);
}
__global__ void performNewIdeaIterationGPU(float* imageOut, float* imageIn, int size, int W, int H) {
    int X = blockIdx.x * blockDim.x + threadIdx.x;
    int Y = blockIdx.y * blockDim.y + threadIdx.y;
    if (X >= W || Y >= H) {
        return;
    }

    float sumR = 0.0;
    float sumG = 0.0;
    float sumB = 0.0;
    int countIncluded = 0;
    for(int x = -size; x <= size; x++) {

        for(int y = -size; y <= size; y++) {
            int currentX = X + x;
            int currentY = Y + y;

            // Check if we are outside the bounds
            if(currentX < 0)
                continue;
            if(currentX >= W)
                continue;
            if(currentY < 0)
                continue;
            if(currentY >= H)
                continue;

            // Now we can begin
            int i = (W * currentY + currentX) * 3;
            sumR += imageIn[i];
            sumG += imageIn[i + 1];
            sumB += imageIn[i + 2];

            // Keep track of how many values we have included
            countIncluded++;
        }

    }
    float valueR = sumR / countIncluded;
    float valueG = sumG / countIncluded;
    float valueB = sumB / countIncluded;

    int i = (W * Y + X) * 3;
    imageOut[i] = valueR;
    imageOut[i + 1] = valueG;
    imageOut[i + 2] = valueB;
}

__global__ void performNewIdeaFinalizationGPU(float* imageInSmall, float* imageInLarge, unsigned char* imageOut, int W, int H) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H) {
        return;
    }
    int i = (W * y + x) * 3;

    float value = (imageInLarge[i] - imageInSmall[i]);
    if(value > 255.0f)
        imageOut[i] = 255;
    else if (value < -1.0f) {
        value = 257.0f+value;
        if(value > 255.0f)
            imageOut[i] = 255;
        else
            imageOut[i] = floorf(value);
    } else if (value > -1.0f && value < 0.0f) {
        imageOut[i] = 0;
    } else {
        imageOut[i] = floorf(value);
    }
    value = (imageInLarge[i+1] - imageInSmall[i+1]);
    if(value > 255.0f)
        imageOut[i+1] = 255;
    else if (value < -1.0f) {
        value = 257.0f+value;
        if(value > 255.0f)
            imageOut[i+1] = 255;
        else
            imageOut[i+1] = floorf(value);
    } else if (value > -1.0f && value < 0.0f) {
        imageOut[i+1] = 0;
    } else {
        imageOut[i+1] = floorf(value);
    }

    value = (imageInLarge[i+2] - imageInSmall[i+2]);
    if(value > 255.0f)
        imageOut[i+2] = 255;
    else if (value < -1.0f) {
        value = 257.0f+value;
        if(value > 255.0f)
            imageOut[i+2] = 255;
        else
            imageOut[i+2] = floorf(value);
    } else if (value > -1.0f && value < 0.0f) {
        imageOut[i+2] = 0;
    } else {
        imageOut[i+2] = floorf(value);
    }
}
int main(int argc, char** argv) {

  PPMImage *image;

  if(argc > 1) {
    image = readPPM("flower.ppm");
  } else {
    image = readStreamPPM(stdin);
  }

  int W = image->x;
  int H = image->y;
  dim3 blockDim(
      ceilf((W + 31) / 32),
      ceilf((H + 31) / 32)
  );
  dim3 threadDim(
      32,
      32
  );

  size_t data_size = 3*W*H*sizeof(float);
  size_t PPMdata_size = sizeof(unsigned char) * W * H;

  float *deviceUnchanged;
  float *deviceBuffer;
  float *deviceSmall;
  float *deviceBig;
  unsigned char *deviceOut;

  PPMImage *imageOut;
  imageOut = (PPMImage *)malloc(sizeof(PPMImage));
  imageOut->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));
  imageOut->x = W;
  imageOut->y = H;
  // Allocate memory on the GPU
  hipMalloc(&deviceOut, PPMdata_size);
  hipMalloc(&deviceUnchanged, data_size);
  hipMalloc(&deviceBuffer, data_size);
  hipMalloc(&deviceSmall, data_size);
  hipMalloc(&deviceBig, data_size);

  hipMemcpy(deviceOut, image->data, PPMdata_size, hipMemcpyHostToDevice);

  convertImageToNewFormatGPU<<<blockDim, threadDim>>>(deviceUnchanged, deviceOut, W, H);

  // Do iterations
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceSmall, deviceUnchanged, 2, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceBuffer, deviceSmall, 2, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceSmall, deviceBuffer, 2, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceBuffer, deviceSmall, 2, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceSmall, deviceBuffer, 2, W, H);

  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceBig, deviceUnchanged, 3, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceBuffer, deviceBig, 3, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceBig, deviceBuffer, 3, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceBuffer, deviceBig, 3, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceBig, deviceBuffer, 3, W, H);


  performNewIdeaFinalizationGPU<<<blockDim, threadDim>>>(deviceSmall, deviceBig, deviceOut, W, H);
  hipMemcpy(imageOut->data, deviceOut, PPMdata_size, hipMemcpyDeviceToHost);
  if(argc > 1) {
    writePPM("flower_tiny.ppm", imageOut);
  } else {
    writeStreamPPM(stdout, imageOut);
  }

  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceSmall, deviceUnchanged, 5, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceBuffer, deviceSmall, 5, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceSmall, deviceBuffer, 5, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceBuffer, deviceSmall, 5, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceSmall, deviceBuffer, 5, W, H);

  performNewIdeaFinalizationGPU<<<blockDim, threadDim>>>(deviceBig, deviceSmall, deviceOut, W, H);
  hipMemcpy(imageOut->data, deviceOut, PPMdata_size, hipMemcpyDeviceToHost);
  if(argc > 1) {
    writePPM("flower_small.ppm", imageOut);
  } else {
    writeStreamPPM(stdout, imageOut);
  }

  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceBig, deviceUnchanged, 8, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceBuffer, deviceBig, 8, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceBig, deviceBuffer, 8, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceBuffer, deviceBig, 8, W, H);
  performNewIdeaIterationGPU<<<blockDim, threadDim>>>(deviceBig, deviceBuffer, 8, W, H);

  performNewIdeaFinalizationGPU<<<blockDim, threadDim>>>(deviceSmall, deviceBig, deviceOut, W, H);
  hipMemcpy(imageOut->data, deviceOut, PPMdata_size, hipMemcpyDeviceToHost);
  if(argc > 1) {
    writePPM("flower_medium.ppm", imageOut);
  } else {
    writeStreamPPM(stdout, imageOut);
  }

  hipFree(deviceUnchanged);
  hipFree(deviceBuffer);
  hipFree(deviceSmall);
  hipFree(deviceBig);
  hipFree(deviceOut);
  free(image->data);
  free(image);
  return 0;
}
