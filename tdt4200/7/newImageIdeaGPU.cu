#include "hip/hip_runtime.h"
#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "ppmCU.h"

// Image from:
// http://7-themes.com/6971875-funny-flowers-pictures.html

typedef struct {
     float red,green,blue;
} AccuratePixel;

typedef struct {
     int x, y;
     AccuratePixel *data;
} AccurateImage;

__global__ void convertImageToNewFormatGPU(int width, int height, float* imageUnchanged, unsigned char* originalData)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= W || y >= H) {
        return;
    }

    int i = y * W * 3 + x * 3;
    imageUnchanged[i] = (float) originalData[i];
    imageUnchanged[i + 1] = (float) originalData[i + 1];
    imageUnchanged[i + 2] = (float) originalData[i + 2];
}

// free memory of an AccurateImage
void freeImage(AccurateImage *image){
	free(image->data);
	free(image);
}

__global__ void performNewIdeaIterationGPU(float *imageOut, float *imageIn, int size, int W, int H) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= W || y >= H) {
      return;
  }
    // For each pixel we compute the magic number
  float sumR = 0;
  float sumG = 0;
  float sumB = 0;
  int countIncluded = 0;
  int senterY = (i+1) / H;
  int senterX = i - W * senterY;

  for(int i = -size; i <= size; i++)
  {
    for(int j = -size; j <= size; j++)
    {
      int currentX = x + i;
      int currentY = y + j;
      // Check if we are outside the bounds
      if(currentX < 0)
        continue;
      if(currentX >= W)
		    continue;
		  if(currentY < 0)
			  continue;
		  if(currentY >= H)
			  continue;

			// Now we can begin
			int offsetOfThePixel = (W * currentY + currentX);
			sumR += imageIn[offsetOfThePixel];
			sumG += imageIn[offsetOfThePixel+1];
			sumB += imageIn[offsetOfThePixel+2];

			// Keep track of how many values we have included
			countIncluded++;
		}
	}
  // Now we compute the final value for all colours
  float valueR = sumR / countIncluded;
  float valueG = sumG / countIncluded;
  float valueB = sumB / countIncluded;

  // Update the output image
  int i = y * W * 3 + x * 3;
  imageOut[i] = valueR;
  imageOut[i + 1] = valueG;
  imageOut[i + 2] = valueB;
}

__global__ void performNewIdeaFinalizationGPU(float* imageInSmall, float* imageInLarge, unsigned char* imageOut, int W, int H) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H)
        return;

    int i = y * W * 3 + x * 3;

    float value = imageInLarge[i] - imageInSmall[i];
    imageOut[i] = (int) value;
    value = imageInLarge[i+1] - imageInSmall[i+1];
    imageOut[i+1] = (int) value;
    value = imageInLarge[i+2] - imageInSmall[i+2];
    imageOut[i+2] = (int) value;
}

int main(int argc, char** argv) {

  PPMImage *image;

  if(argc > 1) {
    image = readPPM("flower.ppm");
  } else {
    image = readStreamPPM(stdin);
  }

  int W = image->x;
  int H = image->y;
  dim3 dimGrid(
      ceilf((width + 31) / 32),
      ceilf((height + 31) / 32)
  );
  dim3 dimBlock(
      32,
      32
  );

  //size_t image_size = sizeof(AccurateImage);
  size_t data_size = 3*W*H*sizeof(float);
  //size_t PPMimage_size = sizeof(PPMImage);
  size_t PPMdata_size = sizeof(PPMPixel) * W * H;

  float *deviceUnchanged;
  float *deviceBuffer;
  float *deviceSmall;
  float *deviceBig;
  float *deviceOut;

  PPMImage *imageOut;
  imageOut = (PPMImage *)malloc(sizeof(PPMImage));
  imageOut->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));

  // Allocate memory on the GPU
  hipMalloc(&deviceOut, data_size);
  hipMemcpy(deviceOut, image->data, PPMdata_size, hipMemcpyHostToDevice);

  hipMalloc(&deviceUnchanged, data_size);

  hipMalloc(&deviceBuffer, data_size);

  hipMalloc(&deviceSmall, data_size);

  hipMalloc(&deviceBig, data_size);

  convertImageToNewFormatGPU<<<numBlocks, numThreads>>>();

  // Do iterations
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceUnchanged, 2, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceSmall, 2, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceBuffer, 2, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceSmall, 2, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceBuffer, 2, W, H);

  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBig, deviceUnchanged, 3, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceBig, 3, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBig, deviceBuffer, 3, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceBig, 3, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBig, deviceBuffer, 3, W, H);


  performNewIdeaFinalizationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceBig, deviceOut, W, H);
  hipMemcpy(imageOut->data, deviceOut, PPMdata_size, hipMemcpyDeviceToHost);
  if(argc > 1) {
    writePPM("flower_medium.ppm", image);
  } else {
    writeStreamPPM(stdout, image);
  }

  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceUnchanged, 5, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceSmall, 5, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceBuffer, 5, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceSmall, 5, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceBuffer, 5, W, H);

  performNewIdeaFinalizationGPU<<<numBlocks, numThreads>>>(deviceBig, deviceSmall, deviceOut, W, H);
  hipMemcpy(imageOut->data, deviceOut, PPMdata_size, hipMemcpyDeviceToHost);
  if(argc > 1) {
    writePPM("flower_medium.ppm", image);
  } else {
    writeStreamPPM(stdout, image);
  }

  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBig, deviceUnchanged, 8, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceBig, 8, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBig, deviceBuffer, 8, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceBig, 8, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBig, deviceBuffer, 8, W, H);

  performNewIdeaFinalizationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceBig, deviceOut, W, H);
  hipMemcpy(imageOut->data, deviceOut, PPMdata_size, hipMemcpyDeviceToHost);
  if(argc > 1) {
    writePPM("flower_medium.ppm", image);
  } else {
    writeStreamPPM(stdout, image);
  }

  hipFree(deviceUnchanged);
  hipFree(deviceBuffer);
  hipFree(deviceSmall);
  hipFree(deviceBig);
  hipFree(deviceOut);
  free(image->data);
  free(image);
  return 0;
}
