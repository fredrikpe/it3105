#include "hip/hip_runtime.h"
#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "ppmCU.h"

// Image from:
// http://7-themes.com/6971875-funny-flowers-pictures.html

typedef struct {
     float red,green,blue;
} AccuratePixel;

typedef struct {
     int x, y;
     AccuratePixel *data;
} AccurateImage;

// Convert a PPM image to a high-precision format
AccurateImage * convertImageToNewFormat(PPMImage *image) {

  AccurateImage *a = (AccurateImage*)malloc(sizeof(AccurateImage));
  a->data = (AccuratePixel*)malloc(image->x*image->y*sizeof(AccuratePixel));
  for(int i = 0; i < image->x*image->y; i++)
  {
		a->data[i].red   = (float) image->data[i].red;
		a->data[i].green = (float) image->data[i].green;
		a->data[i].blue  = (float) image->data[i].blue;
  }
  a->x = image->x;
  a->y = image->y;
  return a;
}

AccurateImage *createEmptyImage(PPMImage *image){
  AccurateImage *imageAccurate;
  imageAccurate = (AccurateImage *)malloc(sizeof(AccurateImage));
  imageAccurate->data = (AccuratePixel*)malloc(image->x * image->y * sizeof(AccuratePixel));
  imageAccurate->x = image->x;
  imageAccurate->y = image->y;

  return imageAccurate;
}

// Convert a high-precision format to a PPM image
PPMImage *convertNewFormatToPPM(AccurateImage *image) {
	// Make a copy
	PPMImage *imagePPM;
	imagePPM = (PPMImage *)malloc(sizeof(PPMImage));
	imagePPM->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));
	for(int i = 0; i < image->x * image->y; i++) {
		imagePPM->data[i].red   = (unsigned char) image->data[i].red;
		imagePPM->data[i].green = (unsigned char) image->data[i].green;
		imagePPM->data[i].blue  = (unsigned char) image->data[i].blue;
	}
	imagePPM->x = image->x;
	imagePPM->y = image->y;

	return imagePPM;
}

// free memory of an AccurateImage
void freeImage(AccurateImage *image){
	free(image->data);
	free(image);
}

__global__ void performNewIdeaIterationGPU(float *imageOut, float *imageIn, int size, int W, int H) {


  // This assumes that (channels * width * height) is divisible by (number of blocks)
  int pixels_per_column = (W*H) / gridDim.x;

  // The start index of a single thread is the start location of the block,
  // plus the thread's index within the block
  int start = (blockIdx.x * pixels_per_column) + threadIdx.x;

  // Run until the start index of the next block
  int stop =  ((blockIdx.x + 1) * pixels_per_column);


  for(int i = start; i < stop; i += blockDim.x) {
    // For each pixel we compute the magic number
    float sumR = 0;
    float sumG = 0;
    float sumB = 0;
    int countIncluded = 0;
    int senterY = (i+1) / H;
    int senterX = i - W * senterY;

    for(int x = -size; x <= size; x++) {

	for(int y = -size; y <= size; y++) {
		int currentX = senterX + x;
		int currentY = senterY + y;
			// Check if we are outside the bounds
			if(currentX < 0)
				continue;
			if(currentX >= W)
				continue;
			if(currentY < 0)
				continue;
			if(currentY >= H)
				continue;

				// Now we can begin
				int numberOfValuesInEachRow = W;
				int offsetOfThePixel = (numberOfValuesInEachRow * currentY + currentX);
				sumR += imageIn[offsetOfThePixel];
				sumG += imageIn[offsetOfThePixel+1];
				sumB += imageIn[offsetOfThePixel+2];

				// Keep track of how many values we have included
				countIncluded++;
			}
		}

		// Now we compute the final value for all colours
		float valueR = sumR / countIncluded;
		float valueG = sumG / countIncluded;
		float valueB = sumB / countIncluded;

		// Update the output image
		int numberOfValuesInEachRow = W; // R, G and B
		int offsetOfThePixel = (numberOfValuesInEachRow * senterY + senterX);
		imageOut[offsetOfThePixel] = valueR;
		imageOut[offsetOfThePixel+1] = valueG;
		imageOut[offsetOfThePixel+2] = valueB;
	}
}

// Perform the final step, and save it as a ppm in imageOut
__global__ void performNewIdeaFinalizationGPU(float *imageInSmall, float *imageInLarge, float *imageOut)
{
  int element_count = sizeof(imageInSmall)/4;
  int pixels_per_column = element_count / gridDim.x;
  int start = (blockIdx.x * pixels_per_column) + threadIdx.x;
  int stop =  ((blockIdx.x + 1) * pixels_per_column);


  for(int i = start; i < stop; i += blockDim.x)
  {
    float value = imageInLarge[i] - imageInSmall[i];
    imageOut[i] = (int) value;
    value = imageInLarge[i+1] - imageInSmall[i+1];
    imageOut[i+1] = (int) value;
    value = imageInLarge[i+2] - imageInSmall[i+2];
    imageOut[i+2] = (int) value;
  }
}

int main(int argc, char** argv) {

  PPMImage *image;

  if(argc > 1) {
    image = readPPM("flower.ppm");
  } else {
    image = readStreamPPM(stdin);
  }

  int W = image->x;
  int H = image->y;
  const int numBlocks = 5;				// Total number of blocks
  const int numThreads = 32;				// Threads per block. 32 is the recommended size by nvidia,

  //size_t image_size = sizeof(AccurateImage);
  size_t data_size = 3*W*H*sizeof(float);
  //size_t PPMimage_size = sizeof(PPMImage);
  size_t PPMdata_size = sizeof(PPMPixel) * W * H;

  AccurateImage *imageUnchanged = convertImageToNewFormat(image); // save the unchanged image from input image
  AccurateImage *imageDummy = createEmptyImage(image);

  float *deviceUnchanged;
  float *deviceBuffer;
  float *deviceSmall;
  float *deviceBig;
  float *deviceOut;

  PPMImage *imageOut;
  imageOut = (PPMImage *)malloc(sizeof(PPMImage));
  imageOut->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));



  // Allocate memory on the GPU
  hipMalloc((void**) &deviceUnchanged, data_size);
  hipMemcpy(deviceUnchanged, imageUnchanged->data, data_size, hipMemcpyHostToDevice);

  hipMalloc((void**) &deviceBuffer, data_size);
  hipMemcpy(deviceBuffer, imageDummy->data, data_size, hipMemcpyHostToDevice);

  hipMalloc((void**) &deviceSmall, data_size);
  hipMemcpy(deviceSmall, imageDummy->data, data_size, hipMemcpyHostToDevice);

  hipMalloc((void**) &deviceBig, data_size);
  hipMemcpy(deviceBig, imageDummy->data, data_size, hipMemcpyHostToDevice);

  hipMalloc((void**) &deviceOut, PPMdata_size);
  hipMemcpy(deviceOut, imageDummy->data, PPMdata_size, hipMemcpyHostToDevice);


  // Do iterations


  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceUnchanged, 2, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceSmall, 2, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceBuffer, 2, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceSmall, 2, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceBuffer, 2, W, H);

  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBig, deviceUnchanged, 3, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceBig, 3, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBig, deviceBuffer, 3, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceBig, 3, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBig, deviceBuffer, 3, W, H);


  performNewIdeaFinalizationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceBig, deviceOut);
  hipMemcpy(image->data, deviceOut, PPMdata_size, hipMemcpyDeviceToHost);
  if(argc > 1) {
    writePPM("flower_medium.ppm", image);
  } else {
    writeStreamPPM(stdout, image);
  }

  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceUnchanged, 5, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceSmall, 5, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceBuffer, 5, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceSmall, 5, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceBuffer, 5, W, H);

  performNewIdeaFinalizationGPU<<<numBlocks, numThreads>>>(deviceBig, deviceSmall, deviceOut);
  //hipMemcpy(image, deviceOut, PPMimage_size, hipMemcpyDeviceToHost);
  hipMemcpy(image->data, deviceOut, PPMdata_size, hipMemcpyDeviceToHost);
  if(argc > 1) {
    writePPM("flower_medium.ppm", image);
  } else {
    writeStreamPPM(stdout, image);
  }

  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBig, deviceUnchanged, 8, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceBig, 8, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBig, deviceBuffer, 8, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBuffer, deviceBig, 8, W, H);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(deviceBig, deviceBuffer, 8, W, H);

  performNewIdeaFinalizationGPU<<<numBlocks, numThreads>>>(deviceSmall, deviceBig, deviceOut);
  //hipMemcpy(image, deviceOut, PPMimage_size, hipMemcpyDeviceToHost);
  hipMemcpy(image->data, deviceOut, PPMdata_size, hipMemcpyDeviceToHost);
  if(argc > 1) {
    writePPM("flower_medium.ppm", image);
  } else {
    writeStreamPPM(stdout, image);
  }

  hipFree(deviceUnchanged);
  hipFree(deviceBuffer);
  hipFree(deviceSmall);
  hipFree(deviceBig);
  hipFree(deviceOut);
  free(image->data);
  free(image);
  return 0;
}
