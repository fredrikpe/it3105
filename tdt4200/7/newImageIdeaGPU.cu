#include "hip/hip_runtime.h"
#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "ppmCU.h"

// Image from:
// http://7-themes.com/6971875-funny-flowers-pictures.html

typedef struct {
     float red,green,blue;
} AccuratePixel;

typedef struct {
     int x, y;
     AccuratePixel *data;
} AccurateImage;

// Convert a PPM image to a high-precision format
__global__ void convertImageToNewFormatGPU(AccurateImage *imageNew, PPMImage *image) {
  int pixels_per_column = element_count / gridDim.x;
  int start = (blockIdx.x * pixels_per_column) + threadIdx.x;
  int stop =  ((blockIdx.x + 1) * pixels_per_column);
  for(int i = start; i < stop; i += blockDim.x)
  {
		imageNew->data[i].red   = (float) image->data[i].red;
		imageNew->data[i].green = (float) image->data[i].green;
		imageNew->data[i].blue  = (float) image->data[i].blue;
	}
}

// Convert a high-precision format to a PPM image
PPMImage *convertNewFormatToPPM(AccurateImage *image) {
	// Make a copy
	PPMImage *imagePPM;
	imagePPM = (PPMImage *)malloc(sizeof(PPMImage));
	imagePPM->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));
	for(int i = 0; i < image->x * image->y; i++) {
		imagePPM->data[i].red   = (unsigned char) image->data[i].red;
		imagePPM->data[i].green = (unsigned char) image->data[i].green;
		imagePPM->data[i].blue  = (unsigned char) image->data[i].blue;
	}
	imagePPM->x = image->x;
	imagePPM->y = image->y;

	return imagePPM;
}

// free memory of an AccurateImage
void freeImage(AccurateImage *image){
	free(image->data);
	free(image);
}

__global__ void performNewIdeaIterationGPU(AccurateImage *imageOut, AccurateImage *imageIn, int size) {
  // This assumes that (channels * width * height) is divisible by (number of blocks)
  int pixels_per_column = element_count / gridDim.x;

  // The start index of a single thread is the start location of the block,
  // plus the thread's index within the block
  int start = (blockIdx.x * pixels_per_column) + threadIdx.x;

  // Run until the start index of the next block
  int stop =  ((blockIdx.x + 1) * pixels_per_column);

  int W = imageIn->x;
  int H = imageIn->y;


  for(int i = start; i < stop; i += blockDim.x) {
		// For each pixel we compute the magic number
		float sumR = 0;
		float sumG = 0;
		float sumB = 0;
		int countIncluded = 0;
    int senterY = (i+1) / H;
    int senterX = i - W * senterY;

		for(int x = -size; x <= size; x++) {

			for(int y = -size; y <= size; y++) {
				int currentX = senterX + x;
				int currentY = senterY + y;

				// Check if we are outside the bounds
				if(currentX < 0)
					continue;
				if(currentX >= W)
					continue;
				if(currentY < 0)
					continue;
				if(currentY >= H)
					continue;

				// Now we can begin
				int numberOfValuesInEachRow = imageIn->x;
				int offsetOfThePixel = (numberOfValuesInEachRow * currentY + currentX);
				sumR += imageIn->data[offsetOfThePixel].red;
				sumG += imageIn->data[offsetOfThePixel].green;
				sumB += imageIn->data[offsetOfThePixel].blue;

				// Keep track of how many values we have included
				countIncluded++;
			}
		}

		// Now we compute the final value for all colours
		float valueR = sumR / countIncluded;
		float valueG = sumG / countIncluded;
		float valueB = sumB / countIncluded;

		// Update the output image
		int numberOfValuesInEachRow = imageOut->x; // R, G and B
		int offsetOfThePixel = (numberOfValuesInEachRow * senterY + senterX);
		imageOut->data[offsetOfThePixel].red = valueR;
		imageOut->data[offsetOfThePixel].green = valueG;
		imageOut->data[offsetOfThePixel].blue = valueB;
	}
}

// Perform the final step, and save it as a ppm in imageOut
__global__ void performNewIdeaFinalizationGPU(AccurateImage *imageInSmall, AccurateImage *imageInLarge, PPMImage *imageOut)
{
  int pixels_per_column = element_count / gridDim.x;
  int start = (blockIdx.x * pixels_per_column) + threadIdx.x;
  int stop =  ((blockIdx.x + 1) * pixels_per_column);

  imageOut->x = imageInSmall->x;
  imageOut->y = imageInSmall->y;

  for(int i = start; i < stop; i += blockDim.x) {
  {
    float value = imageInLarge->data[i].red - imageInSmall->data[i].red;
    imageOut->data[i].red = (int) value;
    value = imageInLarge->data[i].green - imageInSmall->data[i].green;
    imageOut->data[i].green = (int) value;
    value = imageInLarge->data[i].blue - imageInSmall->data[i].blue;
    imageOut->data[i].blue = (int) value;
  }
}

__global__ void fiveIterationsGPU(AccurateImage *imageNew, AccurateImage *imageUnchanged, AccurateImage *imageBuffer, int size)
{
  performNewIdeaIterationGPU<<<5, 32>>>(imageNew, imageUnchanged, size);
  performNewIdeaIterationGPU<<<5, 32>>>(imageBuffer, imageNew, size);
  performNewIdeaIterationGPU<<<5, 32>>>(imageNew, imageBuffer, size);
  performNewIdeaIterationGPU<<<5, 32>>>(imageBuffer, imageNew, size);
  performNewIdeaIterationGPU<<<5, 32>>>(imageNew, imageBuffer, size);
}

int main(int argc, char** argv) {

	PPMImage *image;

	if(argc > 1) {
		image = readPPM("flower.ppm");
	} else {
		image = readStreamPPM(stdin);
	}

  int W = image->x;
  int H = image->y;
  const int numBlocks = 5;				// Total number of blocks
  const int numThreads = 32;				// Threads per block. 32 is the recommended size by nvidia,

  size_t image_size = sizeof(AccurateImage);
  size_t data_size = sizeof(AccuratePixel) * W * H;
  size_t PPMimage_size = sizeof(PPMImage);
  size_t PPMdata_size = sizeof(PPMPixel) * W * H;

  AccurateImage *imageUnchanged;
  AccurateImage *imageBuffer;
  AccurateImage *imageSmall;
  AccurateImage *imageBig;
  PPMImage *imageOut;

  // Allocate memory on the GPU
  hipMalloc((void**) &imageUnchanged, image_size);
  hipMalloc((void**) &imageUnchanged.data, data_size);
  hipMalloc((void**) &imageBuffer, image_size);
  hipMalloc((void**) &imageBuffer.data, data_size);
  hipMalloc((void**) &imageSmall, image_size);
  hipMalloc((void**) &imageSmall.data, data_size);
  hipMalloc((void**) &imageBig, image_size);
  hipMalloc((void**) &imageBig.data, data_size);
  hipMalloc((void**) &imageOut, PPMimage_size);
  hipMalloc((void**) &imageOut.data, PPMdata_size);

  convertImageToNewFormatGPU<<<numBlocks, numThreads>>>(imageUnchanged, image);

  fiveIterationsGPU<<<numBlocks, numThreads>>>(imageSmall, imageUnchanged, imageBuffer, 2);
  fiveIterationsGPU<<<numBlocks, numThreads>>>(imageBig, imageUnchanged, imageBuffer, 3);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(imageSmall, imageBig, imageOut);
  hipMemcpy(image, imageOut, PPMimage_size, hipMemcpyDeviceToHost);
  if(argc > 1) {
    writePPM("flower_medium.ppm", image);
  } else {
    writeStreamPPM(stdout, image);
  }

  fiveIterationsGPU<<<numBlocks, numThreads>>>(imageSmall, imageUnchanged, imageBuffer, 5);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(imageBig, imageSmall, imageOut);
  hipMemcpy(image, imageOut, PPMimage_size, hipMemcpyDeviceToHost);
  if(argc > 1) {
    writePPM("flower_medium.ppm", image);
  } else {
    writeStreamPPM(stdout, image);
  }

  fiveIterationsGPU<<<numBlocks, numThreads>>>(imageBig, imageUnchanged, imageBuffer, 8);
  performNewIdeaIterationGPU<<<numBlocks, numThreads>>>(imageSmall, imageBig, imageOut);
  hipMemcpy(image, imageOut, PPMimage_size, hipMemcpyDeviceToHost);
  if(argc > 1) {
    writePPM("flower_medium.ppm", image);
  } else {
    writeStreamPPM(stdout, image);
  }

  hipFree(imageUnchanged);
  hipFree(imageBuffer);
  hipFree(imageSmall);
  hipFree(imageBig);
  hipFree(imageOut);
  hipFree(imageUnchanged->data);
  hipFree(imageBuffer->data);
  hipFree(imageSmall->data);
  hipFree(imageBig->data);
  hipFree(imageOut->data);
  free(image->data);
  free(image);
	return 0;
}
